// mandelbrot.cu

// Calculate number of iterations required to make each point in Mandelbrot Set diverge and colour the corresponding pixel
// Tends to work faster with floats rather than doubles - but at the expense of "colour blocking" at lower resolutions

// Paul Saunders
// Mercantec
// 03/11-2011


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__device__ inline int calcMandelbrot(const double xPos, const double yPos, const int crunch)
{
    double y = yPos;
    double x = xPos;
    double yy = y * y;
    double xx = x * x;
    int i = crunch;

    while (--i && (xx + yy < 4.0f)) {
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;
    }
    return i;
} // CalcMandelbrot - count down until iterations are used up or until the calculation diverges

/*
__device__ void RGB(int x, int y, unsigned char* m, int step, int iter_count)
{
  unsigned char *p;
  unsigned int rgb;
  p = ((unsigned char *) (m + step*x)+3*y);
  rgb = *p+((*(p+1))<<8)+((*(p+2))<<16);
  
  rgb = iter_count*2048;
  
  *p = (unsigned char) (rgb&0xff);
  *(p+1) = (unsigned char) ((rgb>>8)&0xff);
  *(p+2) = (unsigned char) ((rgb>>16)&0xff);
  return;
} //Use calculated iteration count to determine the colour for each pixel 
*/
__global__ void Count(unsigned int *img, int rows, int cols, int step, int max_iterations, double centre_x, double centre_y, double size, int image_size)
{
  double rowfac = ((double) rows)/gridDim.x;
  int rowstart = blockIdx.x*rowfac;
  int rowend = (blockIdx.x+1)*rowfac;
  double colfac = ((double) cols)/blockDim.x;
  int colstart = threadIdx.x*colfac;
  int colend = (threadIdx.x+1)*colfac;
  double left_edge = centre_x - size/2.0;
  double top_edge = centre_y - size/2.0;
  double pixel_step = size/image_size;
  unsigned int *p;
  for (int i=rowstart; i<rowend; i++)
    {
      for (int j=colstart; j<colend; j++)
      {
	p = (unsigned int*) img + ((step * i) + j);
        *p = (unsigned int) calcMandelbrot(left_edge + j * pixel_step, top_edge + i * pixel_step, max_iterations);
      }
    }
}  //Divide calculations between the requested number of blocks and threads, having used the matrix's geometry to determine the values input to the calculation for each pixel

void startCUDA(int blocks, int threads, int iterations, double centre_x, double centre_y, double size, unsigned int* img, int rows, int cols, int step, int image_size)
{
   if (img!=NULL)
   {
      dim3 dimBlock(threads, 1, 1);
	  dim3 dimGrid(blocks, 1, 1);
      
      unsigned int *CUDAimg;
      hipMalloc((void**) &CUDAimg, rows*cols);
      hipMemcpy(CUDAimg, img, rows*cols, hipMemcpyHostToDevice);
      Count<<<dimGrid, dimBlock>>>(CUDAimg, rows, cols, step, iterations, centre_x, centre_y, size, image_size);
      hipMemcpy(img, CUDAimg, rows*cols, hipMemcpyDeviceToHost);
      hipFree(CUDAimg);
   }
}  // Allocate sufficient memory for the whole image (@3 bytes per pixel), transfer it to the graphics card (host to device), start the calculation process and, when complete, transfer the memory (containing the calculated values) back to the host
