#include "hip/hip_runtime.h"
/*
 * This program computes and displays all or part of the Mandelbrot 
 * set.  By default, it examines all points in the complex plane
 * that have both real and imaginary parts between -2 and 2.  
 * Command-line parameters allow zooming in on a specific part of
 * this range.
 * 
 * Usage:
 *   mandelbrot maxiter [x0 y0 size]
 * where 
 *   maxiter denotes the maximum number of iterations at each point
 *   x0, y0, and size specify the range to examine (a square 
 *     centered at x0 + iy0 of size 2*size by 2*size -- by default, 
 *     a square of size 4 by 4 centered at the origin)
 * 
 * Input:  none, except the optional command-line arguments
 * Output: a graphical display as described in Wilkinson & Allen,
 *   displayed using the X Window system, plus text output to
 *   standard output showing the above parameters, plus execution
 *   time in seconds.
 * 
 * 
 * Code originally code obtained from Web site for Wilkinson and Allen's
 * text on parallel programming:
 * http://www.cs.uncc.edu/~abw/parallel/par_prog/
 * 
 * Reformatted and revised by B. Massingill.
 * Rewritten for Mercantec MPI/CoE Cluster Computing Course by Paul Saunders.
 * 
 * Reformatted and merged with Mandelbrot CUDA by Kim Henriksen
 */
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <omp.h>
#include <X11/Xlib.h>
#include <X11/Xutil.h>
#include <X11/Xos.h>
#define MPICH_SKIP_MPICXX
#include "/usr/include/mpich2/mpi.h"
//#include "/usr/lib/openmpi/include/mpi.h"
/* Functions for GUI */
#include "mandelbrot_guiV1.h"     /* has setup(), interact() */
#include "mandelbrot.h"     /* has setup(), interact() */


/* Default values for things. */
#define N           6          /* size of problem space (x, y from -N to N) */
#define NPIXELS     800         /* size of display window in pixels */
#define FIXED_ZOOM_FACTOR 3.0
#define DATA_TAG 1
#define CUDA_BLOCKS 32
#define CUDA_THREADS 128
#define MBROT_ITER 4096

int master_program(int nWorkers, int width, int height, int subBlockHeight, double real_min, double real_max, double imag_min, double imag_max, int maxiter);
int worker_program(int width, int height, double centre_x, double centre_y, double size, int total_pixel_height);

/* ---- Main program ---- */
int main(int argc, char *argv[]) {
    int nprocs;
    int myid;
    int returnval;

    int maxiter;
    double real_min = -N;
    double real_max = N;
    double imag_min = -N;
    double imag_max = N;
    double size;

    int width = NPIXELS; /* dimensions of display window */
    int height = NPIXELS;
    int centre_x = 0;

    if (MPI_Init(&argc, &argv) != MPI_SUCCESS) {
        fprintf(stderr, "MPI Initialisation Error\n");
        exit(EXIT_FAILURE);
    }
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
    MPI_Comm_rank(MPI_COMM_WORLD, &myid);

    if (nprocs < 2) {
        fprintf(stderr, "Number of processes must be at least 2, but I only have %i\n", nprocs);
        MPI_Finalize();
        exit(EXIT_FAILURE);
    }

    /* Check command-line arguments */
    if ((argc < 3) || ((argc > 3) && (argc < 6))) {
        if (myid == 0) {
            fprintf(stderr, "usage:  %s maxiter block_size [x0 y0 size]\n", argv[0]);
        }
        MPI_Finalize();
        exit(EXIT_FAILURE);
    }

    if ((nprocs - 1) * atoi(argv[2]) > NPIXELS || NPIXELS % ((nprocs - 1) * atoi(argv[2])) != 0) {
        if (myid == 0) {
            fprintf(stderr, "%s%d%s%d%s", "Please ensure that the product of number of worker processes and block size is a divisor of ", NPIXELS, " and that the product is not greater than ", NPIXELS, "\n");
        }
        MPI_Finalize();
        exit(EXIT_FAILURE);
    }

    /* Process command-line arguments */
    maxiter = atoi(argv[1]);
    if (argc > 2) {
        double x0 = atof(argv[2]);
        double y0 = atof(argv[3]);
        size = atof(argv[4]);
        real_min = x0 - size;
        real_max = x0 + size;
        imag_min = y0 - size;
        imag_max = y0 + size;
    }

    // Divide image into chunks, which is equally great, one chunk for each node
    double image_block_size = size / (nprocs - 1);
    double centre_y = imag_min + image_block_size * (2 * myid - 1);

    unsigned int *data_msg = (unsigned int*) malloc((height * width) * sizeof (unsigned int));
    /* Call workers to do calculations, master to collect and display results */
    if (myid == 0) {
        returnval = master_program(nprocs - 1, width, height, image_block_size, real_min, real_max, imag_min, imag_max, maxiter);
    } else {
        returnval = worker_program(width, image_block_size, centre_x, centre_y, size, height);
    }

    /* Finish up */
    MPI_Finalize();

    return returnval;
}


int master_program(int nWorkers, int width, int height, int image_block_size, double real_min, double real_max, double imag_min, double imag_max, int maxiter) {
    Display *display;
    Window win;
    GC gc;
    long min_color, max_color;
    int setup_return;

    int start_row, end_row, current_row, iteration_count, ps;
    double start_time, end_time;

    int *data_msg = (int*) malloc(((image_block_size * width) + 2) * sizeof (int));

    MPI_Status status;

    int col;

    /* Initialize for graphical display */
    setup_return = setup(width, height, &display, &win, &gc, &min_color, &max_color);
    if (setup_return != 1) {
        fprintf(stderr, "Unable to initialize display, continuing\n");
    }
    /* (if not successful, continue but don't display results) */

    // Choose which events we want to handle   
    XSelectInput(display, win, ButtonPressMask | KeyPressMask);

    /*Start timing*/
    start_time = MPI_Wtime();

    /*Receive results from workers and draw points*/
    for (ps = 1; ps <= nWorkers; ps++) {
        MPI_Recv(data_msg, (image_block_size * width) + 2, MPI_INT, MPI_ANY_SOURCE, DATA_TAG, MPI_COMM_WORLD, &status);
        start_row = image_block_size * (nWorkers - 1);
        end_row = start_row + image_block_size - 1;

        for (current_row = start_row; current_row < end_row; current_row++) {
            for (col = 0; col < width; col++) {
                iteration_count = data_msg[((current_row - start_row) * width) + col];

                if (iteration_count < maxiter - 1) {
                    XSetForeground(display, gc, g_mapEntry[iteration_count % NUM_COLORS]);
                    //fprintf(stderr, "Plotting Row\n%d\n", current_row);
                    XDrawPoint(display, win, gc, col, current_row);
                } else {
                    XSetForeground(display, gc, g_mapEntry[0]);
                    XDrawPoint(display, win, gc, col, current_row);
                    //fprintf(stderr, "%d\n", iteration_count);
                }
            }
        }
    }

    /* Be sure all output is written */
    XFlush(display);

    end_time = MPI_Wtime();

    /*Produce text output*/
    double centre_real, centre_imag;
    centre_real = (real_max + real_min) / 2.0;
    centre_imag = (imag_max + imag_min) / 2.0;

    fprintf(stdout, "\n");
    fprintf(stdout, "MPI program\n");
    fprintf(stdout, "Number of worker processes = %d\n", nWorkers);
    fprintf(stdout, "centre = (%g, %g), size = %g\n", centre_real, centre_imag, (real_max - real_min) / 2);
    fprintf(stdout, "Maximum iterations = %d\n", maxiter);
    fprintf(stdout, "Execution Time in seconds = %g\n", end_time - start_time);
    fprintf(stdout, "\n");

    //double scale_real, scale_imag; 
    XEvent report;
    Window root_return, child_return;
    int root_x_return, root_y_return;
    int win_x_return, win_y_return;
    int j;
    unsigned int mask_return;

    //Compute scaling factors (for processing mouse clicks) 
    double scale_real = (double) (real_max - real_min) / (double) width;
    double scale_imag = (double) (imag_max - imag_min) / (double) height;

    //Event loop
    XNextEvent(display, &report);

    switch (report.type) {
        case ButtonPress:
            XQueryPointer(display, win, &root_return, &child_return, &root_x_return, &root_y_return, &win_x_return, &win_y_return, &mask_return);
            centre_real = real_min + ((double) win_x_return * scale_real);
            centre_imag = imag_min + ((double) (height - 1 - win_y_return) * scale_imag);
            fprintf(stderr, "coordinates = (%g, %g)\n", centre_real, centre_imag);
            fflush(stderr);

            fprintf(stderr, "%s\n", "Time to recalculate");
            return 1;

        case KeyPress:

            return 3;

    }

    free(data_msg);

    for (j = 0; j < NUM_COLORS; ++j) {
        XFreeColors(display, DefaultColormapOfScreen(DefaultScreenOfDisplay(display)), &g_mapEntry[j], 1, 0);
    }

    return 0;
}

int worker_program(int width, int height, double centre_x, double centre_y, double size, int total_pixel_height) {
    //unsigned int *img = malloc((height * width) * sizeof (unsigned int));
    unsigned int *data_msg = (unsigned int*) malloc((height * width) * sizeof (unsigned int));

    startCUDA(CUDA_BLOCKS, CUDA_THREADS, MBROT_ITER, centre_x, centre_y, size, data_msg, width, total_pixel_height, width, NPIXELS);
    //data_msg = (int*) img;
    MPI_Send(data_msg, (height * width), MPI_INT, 0, DATA_TAG, MPI_COMM_WORLD);

    return 2;
}
